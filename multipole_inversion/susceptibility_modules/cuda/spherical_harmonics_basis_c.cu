#include "hip/hip_runtime.h"
#include "spherical_harmonics_basis_c.cuh"
#include <cmath>
#include <math.h>
#include <stdio.h>


// Define magnetic constant in GPU
__device__ double Cm = 1e-7;

// The implementation here uses a grid-stride loop:
// https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
__global__ void pop_matrix_dipole(double * Q, double * dip_r, double * pos_r,
                                  unsigned long long Nsources,
                                  unsigned long long Nsensors,
                                  int multipole_order, int n_multipoles,
                                  int verbose) {


    // The thread's unique number
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int threadsInBlock = blockDim.x;

    for (unsigned long long n = global_idx; n < (Nsources * Nsensors); n += stride) {

        unsigned long long i_sensor = n / Nsources;
        unsigned long long i_source = n % Nsources;
        // printf("n = %ld isens = %ld isource = %ld\n", n, i_sensor, i_source);

        double x = pos_r[3 * i_sensor    ] - dip_r[3 * i_source    ];
        double y = pos_r[3 * i_sensor + 1] - dip_r[3 * i_source + 1];
        double z = pos_r[3 * i_sensor + 2] - dip_r[3 * i_source + 2];
        double r2 = x * x + y * y + z * z;
        double r = sqrt(r2);

        // Multipole field susceptibilities; we will re-use this matrix using
        // the largest number of multipoles
        double * p = (double *) malloc(sizeof(double) * (2 * multipole_order + 1));
        int k;
        double f;

        // DIPOLE
        if (multipole_order > 0) {
            f = 1e-7 / (r2 * r2 * r);
            p[2] = (3 * z * z - r2);
            p[1] = (3 * y * z);
            p[0] = (3 * x * z);
            // Assign the 3 dipole entries in the 1st 3 entries of the Q matrix
            for (k = 0; k < 3; ++k) Q[n_multipoles * n + k] = f * p[k];
            for (k = 0; k < 3; ++k) printf("%ld %f\n", n, Q[n_multipoles * n + k]);
        }
        // QUADRUPOLE
        else if (multipole_order > 1) {
            double z2 = z * z;
            // Quad Field from the Cart version of Quad field SHs, by Stone et al
            f = 1e-7 / (r2 * r2 * r2 * r);
            p[0] = sqrt(3 / 2.0) * z * (-3 * r2 + 5 * z2);
            p[1] = -sqrt(2.0) * x * (r2 - 5 * z2);
            p[2] = -sqrt(2.0) * y * (r2 - 5 * z2);
            p[3] = (5 / sqrt(2.0)) * (x * x - y * y) * z;
            p[4] = 5 * sqrt(2.0) * x * y * z;

            for (k = 0; k < 5; ++k) Q[n_multipoles * n + k + 3] = f * p[k];
        }
        // OCTUPOLE
        else if (multipole_order > 2) {
            // Oct Field from the Cartesian version of Octupole field SHs, by Stone et al
            double r4 = r2 * r2;
            double x2 = x * x;
            double y2 = y * y;
            double z2 = z * z;
            f = 1e-7 / (r4 * r4 * r);
            p[0] = (3 * r4 - 30 * r2 * z2 + 35 * (z2 * z2)) / sqrt(10.0);
            p[1] = sqrt(15.0) * x * z * (-3 * r2 + 7 * z2) / 2;
            p[2] = sqrt(15.0) * y * z * (-3 * r2 + 7 * z2) / 2;
            p[3] = -sqrt(1.5) * (x2 - y2) * (r2 - 7 * z2);
            p[4] = -sqrt(6.0) * x * y * (r2 - 7 * z2);
            p[5] = 7 * x * (x2 - 3 * y2) * z / 2;
            p[6] = -7 * y * (-3 * x2 + y2) * z / 2;

            for (k = 0; k < 7; ++k) Q[n_multipoles * n + k + 8] = f * p[k];
        }

        free(p);
    } // end for loop
}

/* Parameters
   ----------
   dip_r
       N * 3 array with dipole positions
   pos_r
       M * 3 array with sensor positions
   Q
       Matrix with the field susceptibilities
   multipole_order
       1 -> dipole, 2 -> quadrupole , ...
 */
void populate_matrix_cuda(double * dip_r,
                          double * pos_r,
                          double * Q,
                          unsigned long long Nsources,
                          unsigned long long Nsensors,
                          int multipole_order
                          ) {

    int verbose = 1;

    // Total number of multipole values
    int n_multipoles = multipole_order * (multipole_order + 2);

    unsigned long long Qsize = n_multipoles * Nsources * Nsensors;
    // Each thread will compute `n_multipoles` elements
    unsigned long long Ndip_x_Nsensor = Nsources * Nsensors;

    size_t Q_bytes = sizeof(double) * Qsize;
    // Manual mem allocation: G in GPU and cuboids_dev in GPU
    double *Q_dev;
    // CUDA_ASSERT(hipMalloc((void**)&Q_dev, Q_bytes));
    // (allocate in GPU if enough memory, see below)
    // hipMalloc((void**)&Q_dev, Q_bytes);

    double *dip_r_dev;
    hipMalloc((void**)&dip_r_dev, sizeof(double) * 3 * Nsources);
    // Copy cuboids array from the host to the GPU
    hipMemcpy(dip_r_dev, dip_r, sizeof(double) * 3 * Nsources, hipMemcpyHostToDevice);

    double *pos_r_dev;
    hipMalloc((void**)&pos_r_dev, sizeof(double) * 3 * Nsensors);
    // Copy cuboids array from the host to the GPU
    hipMemcpy(pos_r_dev, pos_r, sizeof(double) * 3 * Nsensors, hipMemcpyHostToDevice);

    // Launch kernel
    // Quadro RTX 6000: 4608 CUDA Cores
    // More refined matrix allocation of blocks if we use smaller n_threads, e.g. 8
    // Use a N of threads multiple of 32 (multiple of warp size; see docs)
    int n_threads = 256;
    // Determine blocks and grid based on problem size:
    // We will use the number of dipoles and sensors only, Q is larger in size
    int n_blocks = ceil(Ndip_x_Nsensor / (float) n_threads);
    dim3 grid(n_blocks, 1, 1);
    dim3 block(n_threads, 1, 1);
    // TODO: should we use LESS blocks so that threads can compute
    // more efficiently taking advantage of the grid-stride loop ?
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#thread-and-block-heuristics

    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the 
                     // maximum occupancy for a full device launch 
    int gridSize;    // The actual grid size needed, based on input size 
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, 
                                       pop_matrix_dipole, 0, 0); 
    // Round up according to array size 
    gridSize = (Ndip_x_Nsensor + blockSize - 1) / blockSize; 
    printf("Grid size = %d\n", gridSize);
    printf("Min Grid size = %d\n", minGridSize);
    printf("Block size = %d\n", blockSize);

    // Checking available memory in GPU:
    size_t free_byte;
    size_t total_byte;
    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double) free_byte / (1024. * 1024.);
    // Quadro RTX 6000: total mem should be 24220.3125 Mb
    double total_db = (double) total_byte / (1024. * 1024.);
    double used_db = total_db - free_db;
    double Q_size_mb = (double) Q_bytes / (1024. * 1024.);
    double dip_r_size_mb = (double) (3 * Nsources * sizeof(double)) / (1024. * 1024.);

    // if(verbose == 0) {
    printf("------------ Nvidia GPU calculation info ------------\n");
    printf("GPU Memory      (MB): free  = %.4f\n", free_db);
    printf("                      used  = %.4f\n", used_db);
    printf("                      total = %.4f\n", total_db);
    printf("Size of Q       (MB): %.4f\n", Q_size_mb);
    printf("Size of dip_r   (MB): %.4f\n", dip_r_size_mb);
    printf("Blocks grid = %d\n", n_blocks);
    printf("Threads per block = %d\n", n_threads);
    // printf("Sensor Matrix dims (rows x cols) = %d x %d\n", (n_multipoles) * Ndip_x_Nsensor);
    // }

    // Allocate G matrix
    hipMalloc((void**)&Q_dev, Q_bytes);

    // Populate matrix in GPU:
    pop_matrix_dipole<<<grid, block>>>(Q_dev, dip_r_dev, pos_r_dev,
                                       Nsources, Nsensors,
                                       multipole_order, n_multipoles,
                                       verbose);
    hipDeviceSynchronize();

    // Copy Q from the GPU to the host
    hipMemcpy(Q, Q_dev, Q_bytes, hipMemcpyDeviceToHost);

    // for (int k = 0; k < Nsensors; ++k) printf("%d %f\n", k, pos_r[k]);
    // for (int k = 0; k < Ndip_x_Nsensor; ++k) printf("%f\n", Q[k]);

    hipFree(Q_dev);
    hipFree(dip_r_dev);
    hipFree(pos_r_dev);

} // main function
